#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
extern"C"{
#include <ppm.h>
}

// kernel CUDA
__global__ void grayscale(pixel* img_in, pixel* img_out, int cols, int rows){
  // récupération des indices globaux dans la grille 2D pour les
  // dimensions X et Y
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  int x = blockIdx.x*blockDim.x + threadIdx.x;

  // on s'assure de ne pas sortir des limites de l'image
  if (x < cols && y < rows){
    int idx = y*cols+x;
    pixel pix = img_in[idx];
    int gray = 
      0.21f * pix.r + 0.71f * pix.g + 0.07f * pix.b;

    img_out[idx].r = gray;
    img_out[idx].g = gray;
    img_out[idx].b = gray;
    // En tant normal, lorsque on s'appuie sur une bibliothèques avec
    // des types abstraits (ici le type pixel), on utilise les
    // fonctions ou macros de la bibliothèques pour manipuler ce type,
    // par exemple PPM_ASSIGN(img_out[y*cols+x], gray, gray, gray);
  }
  
}

// fonction de lecture d'un fichier ppm (voir utilisation dans main)
pixel * readppm (const char* fname, int* cols, int* rows, pixval* maxval, int* format){
  FILE* img_in;
  
  img_in = fopen(fname, "r");
  ppm_readppminit(img_in, cols, rows, maxval, format);

  pixel* out = (pixel*)malloc ((*cols)*(*rows)*sizeof(pixel));
  int i;
  for (i =0; i < *rows; i++){
    ppm_readppmrow(img_in, out+(i*(*cols)), *cols, *maxval, *format);
  }
  return out;
  
}

// fonction d'écriture d'un fichier ppm (voir utilisation dans main)
void writeppm(const char* fname, pixel* out, int cols, int rows, pixval maxval, int format){
  FILE* img_out;
  img_out = fopen(fname, "w+");
  int i;
  
  ppm_writeppminit(img_out, cols, rows, maxval, format);


  for (i =0; i < rows; i++){
    ppm_writeppmrow(img_out, out+(i*(cols)), cols, maxval, 1);
  }
}


int main(){
  
  pixel *ppm_in, *ppm_out = NULL;
  pixel *d_ppm_in, *d_ppm_out = NULL;

  int cols, rows;
  pixval maxval;
  int format;
  int err;
  
  ppm_in = readppm("lena.ppm", &cols, &rows, &maxval, &format);
  
  long size = cols*rows*sizeof(pixel);  

  ppm_out = (pixel*)malloc(size);
  hipMalloc(&d_ppm_in, size);
  hipMalloc(&d_ppm_out, size);
  
  
  err = hipMemcpy(d_ppm_in, ppm_in, size, hipMemcpyHostToDevice);
  if (err != 0) printf("Error %d\n", err);
  
  // on limite la taille des blocs pour maximiser le parallélisme,
  // ici, un bloc = 16*16 = 256 threads ce qui perme t déjà d'occuper
  // correctement les SM (sur une carte récente, on pourra utiliser
  // des blocs de 32*32), sur une image de 512*512 pixels, on lancera
  // donc au moins 1024 (512*512/256) blocs
  dim3 DimBlock(16, 16,1);
  dim3 DimGrid((rows + DimBlock.x -1)/DimBlock.x, (cols + DimBlock.y -1)/DimBlock.y, 1);
  
  grayscale<<<DimGrid, DimBlock>>>(d_ppm_in, d_ppm_out, cols, rows);
  
  err = hipMemcpy(ppm_out, d_ppm_out, size, hipMemcpyDeviceToHost);
  if (err != 0) {
    printf("Error %d\n", err);
    return err;
  }
  
  writeppm("lena_gray.ppm", ppm_out, cols, rows, maxval, 1);
  
  hipFree(d_ppm_in);
  hipFree(d_ppm_out);
  free(ppm_in);
  free(ppm_out);
  return 0;
}
