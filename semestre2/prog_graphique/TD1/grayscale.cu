#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
extern"C"{
#include <pgm.h>
#include <ppm.h>
}

__global__ void cuda_grayscale(pixel* d_ppm_in, pixel* d_ppm_out, int size){
  // identifiant global du thread dans la grille 1D
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  // on s'assure de ne pas sortir des limites du tableau
  if (tid < size){
    //on effectue le remplacement des couleurs par un niveau de gris
    int gris = ((int)PPM_GETR(d_ppm_in[tid]) + (int)PPM_GETG(d_ppm_in[tid]) + (int)PPM_GETB(d_ppm_in[tid]))/3;
    PPM_PUTR(d_ppm_out[tid], gris);
    PPM_PUTG(d_ppm_out[tid], gris);
    PPM_PUTB(d_ppm_out[tid], gris);
  }
}


pixel * readppm (const char* fname, int* cols, int* rows, pixval* maxval, int* format){
  FILE* img_in;
  img_in = fopen(fname, "r");
  
  ppm_readppminit(img_in, cols, rows, maxval, format);
  
  pixel* out = (pixel*)malloc ((*cols)*(*rows)*sizeof(pixel));
  int i;
  for (i =0; i < *rows; i++){
    ppm_readppmrow(img_in, out+(i*(*cols)), *cols, *maxval, *format);
  }
  return out;
}

void writeppm(const char* fname, pixel* out, int cols, int rows, pixval maxval, int format){
  FILE* img_out;
  img_out = fopen(fname, "w+");
  int i;
  
  ppm_writeppminit(img_out, cols, rows, maxval, format);


  for (i =0; i < rows; i++){
    ppm_writeppmrow(img_out, out+(i*(cols)), cols, maxval, 1);
  }
}




int main(){
  pixel *ppm_in, *ppm_out; // en mémoire CPU
  pixel *d_ppm_in, *d_ppm_out; // en mémoire GPU

  /////////////////////////////////////////////
  //Lire image dans ppm_in
  int *cols;
  int *rows;
  pixval* maxval;
  int *format;
  
  ppm_in = readppm("lena.ppm", cols, rows, maxval, format);
  
  ////////////////////////////////////////////
  //Allocation mémoire
  int size = (*cols)*(*rows)*sizeof(pixel);
  ppm_out = (pixel*)malloc (size);
  hipMalloc((void**)&d_ppm_in, size); 
  hipMalloc((void**)&d_ppm_out, size); 
  
  ////////////////////////////////////////////
  //Copie de ppm_in dans d_ppm_in (en mémoire GPU)
  hipMemcpy(d_ppm_in, ppm_in, size, hipMemcpyHostToDevice);

  ////////////////////////////////////////////
  //Déclaration tailles de grilles et blocs + lancement kernel
  int blockSize = 512;
  int gridSize = ((size + blockSize  - 1) / blockSize);

  cuda_grayscale<<<blockSize, gridSize>>>(d_ppm_in, d_ppm_out, size);
  
  ////////////////////////////////////////////
  //Copie du résultat dans ppm_out
  hipMemcpy(ppm_out, d_ppm_out, size, hipMemcpyDeviceToHost);
    
  ////////////////////////////////////////////
  //Ecriture du fichier ppm
  writeppm("lenagris.ppm", ppm_out, *cols, *rows, *maxval, *format);

  ////////////////////////////////////////////
  //Libération mémoire
  /////////////////////////////////////////
  hipFree(d_ppm_in);
  hipFree(d_ppm_out);

  free(ppm_in);
  free(ppm_out);
  
  return 0;
}

/*
imagemagick
# Transformation d'image en niveaux de gris 

## 1) On utilisera la bibliothèque netpbm pour lire et écrire des
   fichiers ppm (http://netpbm.sourceforge.net/doc/libppm.html).
   Proposez une version CPU (séquentielle) du programme qui transforme
   une image couleur lue dans un fichier ppm en niveaux de gris.

## 2) Proposez une version Cuda du même programme qu'en 1.
En utilisant une grille 1D puis 2D.

## 3) En utilisant des transferts asynchrones.

*/

