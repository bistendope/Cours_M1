#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
extern"C"{
#include <ppm.h>
}
#include <sys/time.h>

// Super macro de mesure du temps
// A NE PAS MODIFIER
#define TIME(fun)							\
  do { struct timeval t1, t2;						\
    gettimeofday(&t1, 0);						\
    fun;								\
    gettimeofday(&t2, 0);						\
    double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0; \
    printf("%s, elapsed time : \033[31;01m%g\033[00m\n", #fun , time);			\
  } while (0)

// A NE PAS MODIFIER
pixel * readppm (const char* fname, int* cols, int* rows, pixval* maxval, int* format){
  FILE* img_in;
  
  img_in = fopen(fname, "r");
  ppm_readppminit(img_in, cols, rows, maxval, format);

  pixel* out = (pixel*)malloc ((*cols)*(*rows)*sizeof(pixel));
  int i;
  for (i =0; i < *rows; i++){
    ppm_readppmrow(img_in, out+(i*(*cols)), *cols, *maxval, *format);
  }
  return out;
  
}

// A NE PAS MODIFIER
void writeppm(const char* fname, pixel* out, int cols, int rows, pixval maxval, int format){
  FILE* img_out;
  img_out = fopen(fname, "w+");
  int i;
  
  ppm_writeppminit(img_out, cols, rows, maxval, format);


  for (i =0; i < rows; i++){
    ppm_writeppmrow(img_out, out+(i*(cols)), cols, maxval, 1);
  }
}



/********** A partir d'ici c'est à vous de jouer **********/

/* On commence doucement, sans textures!  Vous pouvez vous appuyer sur
   l'exercice du TD précédent (correction sur Celene) */

// Noyau qui floute une image en utilisant un masque de "rayon" paramétrable
__global__ void blur(pixel* img_in, pixel* img_out, int cols, int rows, int ray){
  // récupération des indices globaux dans la grille 2D pour les
  // dimensions X et Y
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int x = blockIdx.x*blockDim.x + threadIdx.x;

  // on s'assure de ne pas sortir des limites de l'image
  if (x < cols && y < rows){
  int idx, sommeR, sommeG, sommeB;
  int cpt = 0;
  sommeR=0;
  sommeG=0;
  sommeB=0;
  pixel pix;
    for(int xInd = max(0,x-ray); xInd < min(rows, x+ray); xInd++){
    	for(int yInd = max(0, y-ray); yInd < min(cols, y + ray); yInd++){
    	    cpt++;
	    idx = xInd*rows+yInd;
	    pix = img_in[idx];
	    sommeR += pix.r;
	    sommeG += pix.g;
	    sommeB += pix.b;
	}
    }
    idx = x*rows+y;
    img_out[idx].r = sommeR/cpt;
    img_out[idx].g = sommeG/cpt;
    img_out[idx].b = sommeB/cpt;
    // En tant normal, lorsque on s'appuie sur une bibliothèques avec
    // des types abstraits (ici le type pixel), on utilise les
    // fonctions ou macros de la bibliothèques pour manipuler ce type,
    // par exemple PPM_ASSIGN(img_out[y*cols+x], gray, gray, gray);
  }
}

// Fonction CPU qui utilise  le noyau blur
void blur (pixel* ppm_in, pixel* ppm_out, size_t size, int cols, int rows, pixval maxval, int ray) {
  pixel *d_ppm_in, *d_ppm_out = NULL;

  dim3 DimBlock(16, 16,1);
  dim3 DimGrid((rows + DimBlock.x -1)/DimBlock.x, (cols + DimBlock.y -1)/DimBlock.y, 1);
  
  hipMalloc(&d_ppm_in, size);
  hipMalloc(&d_ppm_out, size);
  hipMemcpy(d_ppm_in, ppm_in, size, hipMemcpyHostToDevice);
  blur<<<DimGrid, DimBlock>>>(d_ppm_in, d_ppm_out, cols, rows, ray);
  hipMemcpy(ppm_out, d_ppm_out, size, hipMemcpyDeviceToHost);
  hipFree(d_ppm_in);
  hipFree(d_ppm_out);
}


/* Quand la version "classique" fonctionne, vous pouvez passer à
   l'utilisation de textures */


// Texture globale (pas besoin de la passer en paramère d'un kernel)
// 2D contenant des unsigned int
// A NE PAS MODIFIER 
texture<unsigned int, 2, hipReadModeElementType> tex;


// Même chose qu'avant mais avec une texture.
//
// Vous utiliserez tex2D(texture, x, y)
// pour lire une valeur dans la texture
__global__ void blur_tex(pixel* img_out, int cols, int rows, int ray){
	
  
}



/* Fonction qui lance le noyau blur_tex Pour rappel, pour utiliser une
   texture, il faut :

   + décrire un cudaChennelFormatDesc (ici, on associe notre texture à
   de simples unsigned int, on pourra donc utiliser le code suivant :
   hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindUnsigned);   

   + il faut spécifier les propriétés de notre texture, en particulier:
     ++ son mode de gestion des frontières :
         tex.adressMode[0] pour la dimension 1
         tex.adressMode[1] pour la dimension 2
	ici, on utilisera le mode Wrap (voir API CUDA)
     ++ son mode de filtrage, filterMode, ici on restera sur un mode "non" filtré, 
        en utilisant hipFilterModePoint
     ++ si la texture est normalisée ou non (ici, non)

   + on doit associer la texture à un hipArray, pour cela il faut : 
     ++ allouer un hipArray (voir API CUDA)
     ++ copier les données dans le hipArray sur le GPU (voir API CUDA)
     ++ lier la texture au hipArray avec hipBindTextureToArray (voir API CUDA)

 N'oubliez pas de lancer le kernel, allouer la mémoire pour le résultat etc

*/
void blur_tex (pixel* ppm_in, pixel* ppm_out, size_t size, int cols, int rows, pixval maxval, int ray) {
}
 


/***********   FIN DU TRAVAIL POUR VOUS **********/

// A NE PAS MODIFIER
// le programme utilisera lena.ppm par défaut mais
// vous pouvez lui passer une image ppm (sans l'extension) en
// paramètre pour l'essayer sur d'autres images.
// par exemple :
// ./blur mandril 
int main(int argc, char* argv[]){

  char* name = (argc <= 1)?((char*)"lena"):argv[1];
  char *in = (char*)malloc(sizeof(char)); 
  in = strcat(strcat(in,name), ".ppm");

  char *out = (char*)malloc(sizeof(char)); 
  out = strcat(strcat(out,name), "_blur.ppm");
  
  char *out_tex = (char*)malloc(sizeof(char)); 
  out_tex =  strcat(strcat(out_tex,name), "_blur_tex.ppm");

  printf("Will work on %s and generate %s and %s\n", in, out, out_tex);
  

  pixel *ppm_in, *ppm_out = NULL;
  int cols, rows;
  pixval maxval;
  int format;
  
  ppm_in = readppm(in, &cols, &rows, &maxval, &format);

  long size = cols*rows*sizeof(pixel);  
  ppm_out = (pixel*)malloc(size);
  
  int ray = 25;
  
  TIME(blur(ppm_in, ppm_out, size, cols, rows, maxval, ray));	 
  writeppm(out, ppm_out, cols, rows, maxval, 1);

  
  TIME(blur_tex(ppm_in, ppm_out, size, cols, rows, maxval, ray));	  
  writeppm(out_tex, ppm_out, cols, rows, maxval, 1);

 
  
  free(ppm_in);
  free(ppm_out);
  return 0;
}
