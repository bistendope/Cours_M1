#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(float *a, float *b, float *c, int n)
{

}

int main( int argc, char* argv[] )
{
  // Size of vectors
  int n = 100000;

  // Host input vectors
  float *h_a;
  float *h_b;
  //Host output vector
  float *h_c;

  // Device input vectors
  float *d_a;
  float *d_b;
  //Device output vector
  float *d_c;

  // Size, in bytes, of each vector
  size_t bytes = n*sizeof(float);


  //////////////////////////////////////////
  // Allocate memory for each vector on host

  /////////////////////////////////////////
  // Allocate memory for each vector on GPU

  int i;
  // Initialize vectors on host
  for( i = 0; i < n; i++ ) {
    h_a[i] = sin(i)*sin(i);
    h_b[i] = cos(i)*cos(i);
  }

  /////////////////////////////////////////
  // Copy host vectors to device
  // Use hipMemcpy...


  int blockSize, gridSize;

  /////////////////////////////////////////
  // Number of threads in each thread block
  blockSize = ??;

  ////////////////////////////////////////
  // Number of thread blocks in grid
  gridSize = ??;


  ///////////////////////////////////////
  // Execute the kernel
  vecAdd<<<??, ??>>>(d_a, d_b, d_c, n);

  ///////////////////////////////////////
  // Copy array back to host
  // Use hipMemcpy

  // Sum up vector c and print result divided by n, this should equal 1 within error
  float sum = 0;
  for(i=0; i<n; i++)
    sum += h_c[i];
  printf("final result: %f\n", sum/n);

  /////////////////////////////////////////
  // Release device memory

  ////////////////////////////////////////
  // Release host memory

  return 0;
}
