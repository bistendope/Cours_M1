// pour compiler : nvcc vecAdd.cu -o vecAdd

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(float *a, float *b, float *c, int n){
  // identifiant global du thread dans la grille 1D
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  // on s'assure de ne pas sortir des limites des tableaux a,b,c
  if (tid < n){
    //on effectue une addition élémentaire par thread
    c[tid] = a[tid] + b[tid];
  }
}

int main( int argc, char* argv[] )
{
  // Size of vectors
  int n = 100000;

  // Host input vectors
  float *h_a;
  float *h_b;
  //Host output vector
  float *h_c;

  // Device input vectors
  float *d_a;
  float *d_b;
  //Device output vector
  float *d_c;

  // Size, in bytes, of each vector
  size_t size = n*sizeof(float);


  //////////////////////////////////////////
  // Allocate memory for each vector on host
  h_a = (float*) malloc (size);
  h_b = (float*) malloc (size);
  h_c = (float*) malloc (size);
  

  
  /////////////////////////////////////////
  // Allocate memory for each vector on GPU
  hipMalloc((void**)&d_a, size);
  hipMalloc((void**)&d_b, size);
  hipMalloc((void**)&d_c, size);
  
  int i;
  // Initialize vectors on host
  for( i = 0; i < n; i++ ) {
    h_a[i] = sin(i)*sin(i);
    h_b[i] = cos(i)*cos(i);
  }

  /////////////////////////////////////////
  // Copy host vectors to device
  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);


  int blockSize, gridSize;

  /////////////////////////////////////////
  // Number of threads in each thread block
  blockSize = 512;

  ////////////////////////////////////////
  // Number of thread blocks in grid
  gridSize = (n + blockSize  - 1) / blockSize;


  ///////////////////////////////////////
  // Launch the kernel
  vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

  ///////////////////////////////////////
  // Copy array back to host
  hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

  // Sum up vector c and print result divided by n, this should equal 1 within error
  float sum = 0;
  for(i=0; i<n; i++)
    sum += h_c[i];
  printf("final result: %f\n", sum/n);

  /////////////////////////////////////////
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  ////////////////////////////////////////
  // Release host memory
  free(h_a);
  free(h_b);
  free(h_c);

  return 0;
}
