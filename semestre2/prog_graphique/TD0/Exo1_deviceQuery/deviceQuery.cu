#include <stdio.h>

//mdp: 

// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
  printf("Major revision number:         %d\n",  devProp.major);
  printf("Minor revision number:         %d\n",  devProp.minor);
  printf("Name:                          %s\n",  devProp.name);
  printf("Total global memory:           %zu\n",  devProp.totalGlobalMem);
  printf("Total shared memory per block: %zu\n",  devProp.sharedMemPerBlock);
  printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
  printf("Warp size:                     %d\n",  devProp.warpSize);
  printf("Maximum memory pitch:          %zu\n",  devProp.memPitch);
  printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
  for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
  for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
  printf("Clock rate:                    %d\n",  devProp.clockRate);
  printf("Total constant memory:         %zu\n",  devProp.totalConstMem);
  printf("Texture alignment:             %zu\n",  devProp.textureAlignment);
  printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
  printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
  printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
  return;
}

int main()
{
  // Number of CUDA devices
  int devCount;
  hipGetDeviceCount(&devCount);
  printf("CUDA Device Query...\n");
  printf("There are %d CUDA devices.\n", devCount);

  // Iterate through devices
  for (int i = 0; i < devCount; ++i)
    {
      // Get device properties
      printf("\nCUDA Device #%d\n", i);
      hipDeviceProp_t devProp;
      hipGetDeviceProperties(&devProp, i);
      printDevProp(devProp);
    }

  printf("\nPress any key to exit...");
  char c;
  scanf("%c", &c);

  return 0;
}

/*                                                                                                                 
 Quelques questions à se poser :                                                                                   
                                                                                                                   
+ Quelle est la compute capability de la carte graphique ?      
                                                   
+ Quelles sont les  dimensions maximales d'un bloc (en X, en Y, en Z)?

+ Supposons que nous lançons une grille de blocs unidimensionelle (seulement sur X). 
Si la dimension maximale de la grille est de 65525 sur notre matériel et celle d'un 
bloc de 1024, quel est le nombre maxmal de threads que l'on peut lancer sur notre GPU?
	-1024*65525
	
+ Sous quelles conditions un programmeur pourrait choisir de ne pas utiliser ce nombre
maximum de threads?      
	- On n'en a pas forcément besoin d'autant
	- Si ça ne tombe pas juste
+ D'après vous, qu'est-ce qui peut empêcher un programme de lancer le nombre maximal
de threads sur un GPU?
    
+ Le parallélisme dynamique est-il supporté sur votre carte graphique?
	- yes (Titan et Titan X, 970M également)
*/
